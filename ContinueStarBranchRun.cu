#include "hip/hip_runtime.h"
/*
nvcc StarBranchRun.cu -o ContinueStarBranchRun -lglut -lGL -lGLU -lm
nvcc StarBranchRun.cu -o ContinueStarBranchRun -lglut -lGL -lGLU -lm --use_fast_math
*/


#include "binaryStar.h"

#define BLOCKSIZE 512

//Globals to hold positions, velocities, and forces on both the GPU and CPU
float4 *PlaceHolder; //needs to be hard defined for cuda
float4 *Pos, *Vel, *Force;
float4 *PosGPU, *VelGPU, *ForceGPU;

//Globals to setup the kernals
dim3 BlockConfig, GridConfig;

//Root folder to containing the stars to work with.
char RootFolderName[256] = "";

//Globals read in from the BiuldSetup file
double Pi;
double UniversalGravity;			//Universal gravitational constant in kilometersE3 kilogramsE-1 and secondsE-2 (??? source)
double MassOfSun;
double RadiusOfSun;
double FractionSunMassOfStar1;
double FractionSunMassOfStar2;
double DensityOfHydrogenGas;		//Density of hydrogen gas in kilograms meterE-3 (??? source)
float KH;							//Push back of hygrogen gas element
float KRH;							//Push back reduction of hygrogen gas element
int NumberOfElements;
float Dt;


//Globals to be set by the setRunParameters function
double SystemLengthConverterToKilometers;
double SystemMassConverterToKilograms;
double SystemTimeConverterToSeconds;
int NumberOfElementsInStar1;
int NumberOfElementsInStar2; 

//Globals read in from the BranchSetup file.
float BranchRunTime;
float GrowStartTime;
float GrowStopTime;
float DeltaForceIncrease;
int RecordRate;
int DrawRate;

//File to hold the position and velocity outputs to make videos and analysis of the run.
FILE *PosAndVelFile;

void readBuildParameters()
{
	ifstream data;
	string name;
	float temp;
	
	data.open("BuildSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> Pi;
		getline(data,name,'=');
		data >> UniversalGravity;
		getline(data,name,'=');
		data >> RadiusOfSun;
		getline(data,name,'=');
		data >> MassOfSun;
		
		getline(data,name,'=');
		data >> FractionSunMassOfStar1;
		getline(data,name,'=');
		data >> FractionSunMassOfStar2;
		
		getline(data,name,'=');
		data >> DensityOfHydrogenGas;
		getline(data,name,'=');
		data >> KH;
		getline(data,name,'=');
		data >> KRH;
		
		getline(data,name,'=');
		data >> NumberOfElements;
		
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		getline(data,name,'=');
		data >> temp;
		
		getline(data,name,'=');
		data >> temp;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> temp;
	}
	else
	{
		printf("\nTSU Error could not open run or root Setup file\n");
		exit(0);
	}
	data.close();
}

void generateRunParameters()
{
	double massOfStar1;
	double massOfStar2;
	double massOfHydrogenGasElement;
	double baseDiameterOfHydrogenGasElement;
	
	massOfStar1 = MassOfSun*FractionSunMassOfStar1;
	massOfStar2 = MassOfSun*FractionSunMassOfStar2;
	
	//The mass of an element is just the total mass divided by the number of elements used.
	massOfHydrogenGasElement = (massOfStar1 + massOfStar2)/((double)NumberOfElements);
	
	//We will use the mass of a hydrogen gas element as one unit of mass. 
	//The following constant will convert system masses up to kilograms by multipling 
	//or convert kilograms down to system units by dividing.
	SystemMassConverterToKilograms = massOfHydrogenGasElement;
	
	NumberOfElementsInStar1 = massOfStar1/massOfHydrogenGasElement;
	NumberOfElementsInStar2 = NumberOfElements - NumberOfElementsInStar1;
	
	baseDiameterOfHydrogenGasElement = pow((6.0*massOfHydrogenGasElement)/(Pi*DensityOfHydrogenGas), (1.0/3.0));
	
	//We will use the diameter of a hydrogen gas element as one unit of length. 
	//The following constant will convert system lengths up to kilometers by multipling 
	//or convert kilometers down to system units by dividing.
	SystemLengthConverterToKilometers = baseDiameterOfHydrogenGasElement;
	
	//We will use a time unit so that the universal gravitational constant will be 1. 
	//The following constant will convert system times up to seconds by multipling 
	//or convert seconds down to system units by dividing. Make sure UniversalGravity is fed into the program in kilograms kilometers and seconds!
	SystemTimeConverterToSeconds = sqrt(pow(SystemLengthConverterToKilometers,3)/(SystemMassConverterToKilograms*UniversalGravity));
	
	KH *= SystemTimeConverterToSeconds*SystemTimeConverterToSeconds*SystemLengthConverterToKilometers/SystemMassConverterToKilograms;
	printf("\n KH = %f\n", KH);
}

void readBranchParametersAndSetInitialConditions()
{
	ifstream data;
	string name;
	
	float3 initailPos1, initailPos2, initailVel1, initailVel2;
	
	data.open("BranchSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> initailPos1.x;
		getline(data,name,'=');
		data >> initailPos1.y;
		getline(data,name,'=');
		data >> initailPos1.z;
		
		getline(data,name,'=');
		data >> initailPos2.x;
		getline(data,name,'=');
		data >> initailPos2.y;
		getline(data,name,'=');
		data >> initailPos2.z;
		
		getline(data,name,'=');
		data >> initailVel1.x;
		getline(data,name,'=');
		data >> initailVel1.y;
		getline(data,name,'=');
		data >> initailVel1.z;
		
		getline(data,name,'=');
		data >> initailVel2.x;
		getline(data,name,'=');
		data >> initailVel2.y;
		getline(data,name,'=');
		data >> initailVel2.z;
		
		getline(data,name,'=');
		data >> BranchRunTime;
		
		getline(data,name,'=');
		data >> GrowStartTime;
		
		getline(data,name,'=');
		data >> GrowStopTime;
		
		getline(data,name,'=');
		data >> DeltaForceIncrease;
		
		getline(data,name,'=');
		data >> RecordRate;
		
		getline(data,name,'=');
		data >> DrawRate;
		
		/*
		printf("\n BranchRunTime = %f\n", BranchRunTime);
		printf("\n GrowStartTime = %f\n", GrowStartTime);
		printf("\n GrowStopTime = %f\n", GrowStopTime);
		printf("\n DeltaForceIncrease = %f\n", DeltaForceIncrease);
		printf("\n RecordRate = %d\n", RecordRate);
		printf("\n drawRate = %d\n", DrawRate);
		*/
		
		// Taking the run times into our units,
		BranchRunTime *= (60.0*60.0*24.0)/SystemTimeConverterToSeconds;
		GrowStartTime *= (60.0*60.0*24.0)/SystemTimeConverterToSeconds;
		GrowStopTime *= (60.0*60.0*24.0)/SystemTimeConverterToSeconds;
		
		/*
		printf("\n BranchRunTime = %f\n", BranchRunTime);
		printf("\n GrowStartTime = %f\n", GrowStartTime);
		printf("\n GrowStopTime = %f\n", GrowStopTime);
		*/
		
		for(int i = 0; i < NumberOfElementsInStar1; i++)	
		{
			Pos[i].x += initailPos1.x/SystemLengthConverterToKilometers;
			Pos[i].y += initailPos1.y/SystemLengthConverterToKilometers;
			Pos[i].z += initailPos1.z/SystemLengthConverterToKilometers;
			
			Vel[i].x += initailVel1.x*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
			Vel[i].y += initailVel1.y*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
			Vel[i].z += initailVel1.z*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
		}
		
		for(int i = NumberOfElementsInStar1; i < NumberOfElements; i++)	
		{
			Pos[i].x += initailPos2.x/SystemLengthConverterToKilometers;
			Pos[i].y += initailPos2.y/SystemLengthConverterToKilometers;
			Pos[i].z += initailPos2.z/SystemLengthConverterToKilometers;
			
			Vel[i].x += initailVel2.x*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
			Vel[i].y += initailVel2.y*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
			Vel[i].z += initailVel2.z*SystemTimeConverterToSeconds/SystemLengthConverterToKilometers;
		}
	}
	else
	{
		printf("\nTSU Error could not open run or root Setup file\n");
		exit(0);
	}
	data.close();
}

void errorCheck(const char *message)
{
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess)
  {
    printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

void allocateMemory()
{
	PlaceHolder = (float4*)malloc(NumberOfElements*sizeof(float4));
	Pos = (float4*)malloc(NumberOfElements*sizeof(float4));
	Vel = (float4*)malloc(NumberOfElements*sizeof(float4));
	Force = (float4*)malloc(NumberOfElements*sizeof(float4));
	
	hipMalloc((void**)&PosGPU, NumberOfElements *sizeof(float4));
	errorCheck("hipMalloc Pos");
	hipMalloc((void**)&VelGPU, NumberOfElements *sizeof(float4));
	errorCheck("hipMalloc Vel");
	hipMalloc((void**)&ForceGPU, NumberOfElements *sizeof(float4));
	errorCheck("hipMalloc Force");
	
	PosAndVelFile = fopen("PosAndVel", "wb");
}

void cleanUp()
{
	free(PlaceHolder);
	free(Pos);
	free(Vel);
	free(Force);
	
	hipFree(PosGPU);
	hipFree(VelGPU);
	hipFree(ForceGPU);
	
	fclose(PosAndVelFile);
}

void readInTheInitialsStars()
{
	FILE *temp = fopen("StartPosAndVel","rb");
	fread(Pos, sizeof(float4), NumberOfElements, temp);
	fread(Vel, sizeof(float4), NumberOfElements, temp);
	fclose(temp);
	
	printf("\n************************************************** The stars have been read in\n");
}

void deviceSetup()
{
	if(NumberOfElements%BLOCKSIZE != 0)
	{
		printf("\nTSU Error: Number of Particles is not a multiple of the block size \n\n");
		exit(0);
	}
	
	BlockConfig.x = BLOCKSIZE;
	BlockConfig.y = 1;
	BlockConfig.z = 1;
	
	GridConfig.x = (NumberOfElements-1)/BlockConfig.x + 1;
	GridConfig.y = 1;
	GridConfig.z = 1;
}

__device__ float4 calculateElementElementForce(float4 posMe, float4 posYou, float4 velMe, float4 velYou, float KH, float KRH, float forceGrowth, float pi)
{
	float4 dp, dv, force;
	float r, r2, r3, invr, inOut, force_mag;
	
	dp.x = posYou.x - posMe.x;
	dp.y = posYou.y - posMe.y;
	dp.z = posYou.z - posMe.z;
	r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
	r = sqrt(r2);
	r3 = r2*r;
	
	if(1.0f <= r)
	{
		invr = 1.0f/r3;
		
		force.x = (dp.x*invr);
		force.y = (dp.y*invr);
		force.z = (dp.z*invr);
	}
	else if(0.0f < r)
	{
		invr = 1.0f/r;
		
		dv.x = velYou.x - velMe.x;
		dv.y = velYou.y - velMe.y;
		dv.z = velYou.z - velMe.z;
		inOut = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
		
		if(inOut <= 0) 	force_mag  = 1.0f - (forceGrowth+KH)*pi*(1.0f/6.0f + r3/3.0f - r2/2.0f);
		else 			force_mag  = 1.0f - KRH*(forceGrowth+KH)*pi*(1.0f/6.0f + r3/3.0f - r2/2.0f);
		
		force.x = (dp.x*invr)*force_mag;
		force.y = (dp.y*invr)*force_mag;
		force.z = (dp.z*invr)*force_mag;
	}
	else // Hopefully this line of code never gets reached.
	{
		dv.x = velYou.x - velMe.x;
		dv.y = velYou.y - velMe.y;
		dv.z = velYou.z - velMe.z;
		if(0.0f < (dv.x + dv.y + dv.z)) // Hopefully if it have some velocity it will drift past setting right on top of eachother.
		{
			force.x = 0.0f;
			force.y = 0.0f;
			force.z = 0.0f;
		}
		else // If they have no velocity we will have to kick them off of this position. This will be an unnatural force in the x direction
		{
			force.x = KH;
			force.y = 0.0f;
			force.z = 0.0f;
		}
	}
	return(force);
}


__global__ void getForce(float4 *pos, float4 *vel, float4 *force, float KH, float KRH, float forceGrowth, int NumberOfElementsInStar1, float pi)
{
	int id, ids, i, j;
	float4 posMe, velMe;
	float4 elementElementForce, forceSum;
	
	__shared__ float4 shPos[BLOCKSIZE];
	__shared__ float4 shVel[BLOCKSIZE];

	id = threadIdx.x + blockDim.x*blockIdx.x;
		
	forceSum.x = 0.0f;
	forceSum.y = 0.0f;
	forceSum.z = 0.0f;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	
	velMe.x = vel[id].x;
	velMe.y = vel[id].y;
	velMe.z = vel[id].z;
	
	for(j = 0; j < gridDim.x; j++)
	{
		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x] = vel[threadIdx.x + blockDim.x*j];
		__syncthreads();
	   
		#pragma unroll 32
		for(i = 0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
			if(id != ids)
			{
				if(id < NumberOfElementsInStar1 && ids < NumberOfElementsInStar1)
				{
					elementElementForce = calculateElementElementForce(posMe, shPos[i], velMe, shVel[i], KH, KRH, forceGrowth, pi);
				}
				else
				{
					elementElementForce = calculateElementElementForce(posMe, shPos[i], velMe, shVel[i], KH, KRH, 0.0f, pi);
				}
				forceSum.x += elementElementForce.x;
				forceSum.y += elementElementForce.y;
				forceSum.z += elementElementForce.z;
			}
		}
		__syncthreads();
	}
	force[id].x = forceSum.x;
	force[id].y = forceSum.y;
	force[id].z = forceSum.z;
}

__global__ void moveBodies(float4 *pos, float4 *vel, float4 *force, float dt)
{
	int id;
	
    id = threadIdx.x + blockDim.x*blockIdx.x;

	// There is no need to divide by mass in the velocity lines because the mass of an element is 1.
	vel[id].x += (force[id].x)*dt;
	vel[id].y += (force[id].y)*dt;
	vel[id].z += (force[id].z)*dt;

	pos[id].x += vel[id].x*dt;
	pos[id].y += vel[id].y*dt;
	pos[id].z += vel[id].z*dt;
}

void drawPicture(float4 *pos, int NumberOfElementsInStar1, int NumberOfElementsInStar2)
{	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	
	glBegin(GL_POINTS);
 		for(int i = 0; i < NumberOfElementsInStar1; i++)
		{
			glColor3d(1.0,1.0,0.0);
			glVertex3f(pos[i].x, pos[i].y, pos[i].z);
		}
		
		for(int i = NumberOfElementsInStar1; i < NumberOfElements; i++)
		{
			glColor3d(1.0,0.5,0.0);
			glVertex3f(pos[i].x, pos[i].y, pos[i].z);
		}
	glEnd();
	
	glutSwapBuffers();
}

void recordPosAndVel()
{
	fwrite(Pos, sizeof(float4), NumberOfElements, PosAndVelFile);
	fwrite(Vel, sizeof(float4), NumberOfElements, PosAndVelFile);
}

void recordFinalPosVelStars(float time)
{
	const char *folderName;
	stringstream streamTime;
	string stringTime;
	string stringFolderName;
	
	streamTime << time;
	stringTime = streamTime.str();
	stringFolderName = "PosAndVelAt" + stringTime + "Days";
	folderName = stringFolderName.c_str();
	
	FILE *finalPosAndVelFile;
	
	//finalPosAndVelFile = fopen("FinalPosAndVel", "wb");
	finalPosAndVelFile = fopen(folderName, "wb");
	hipMemcpy( Pos, PosGPU, NumberOfElements *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Pos");
	hipMemcpy( Vel, VelGPU, NumberOfElements *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Vel");
	
	fwrite(Pos, sizeof(float4), NumberOfElements, finalPosAndVelFile);
	fwrite(Vel, sizeof(float4), NumberOfElements, finalPosAndVelFile);
	
	fclose(finalPosAndVelFile);
}

void recordStarStats()
{
	FILE *starStatsFile;
	double massStar1, massStar2;
	double radiusStar1, radiusStar2;
	double densityStar1, densityStar2;
	double radius;
	
	hipMemcpy( Pos, PosGPU, NumberOfElements *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Pos");
	hipMemcpy( Vel, VelGPU, NumberOfElements *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Vel");
	
	massStar1 = NumberOfElementsInStar1*SystemMassConverterToKilograms;
	massStar2 = NumberOfElementsInStar2*SystemMassConverterToKilograms;
	
	radiusStar1 = 0.0;
	for(int i = 0; i < NumberOfElementsInStar1; i++)
	{
		radius = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
		if(radiusStar1 < radius) radiusStar1 = radius;
	}
	
	radiusStar2 = 0.0;
	for(int i = NumberOfElementsInStar1; i < NumberOfElements; i++)
	{
		radius = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
		if(radiusStar2 < radius) radiusStar2 = radius;
	}
	
	radiusStar1 *= SystemLengthConverterToKilometers;
	radiusStar2 *= SystemLengthConverterToKilometers;
	
	densityStar1 = massStar1/((4.0/3.0)*Pi*radiusStar1*radiusStar1*radiusStar1);
	densityStar2 = massStar2/((4.0/3.0)*Pi*radiusStar2*radiusStar2*radiusStar2);
	
	starStatsFile = fopen("StarStats", "wb");
		fprintf(starStatsFile, " The conversion parameters to take you to and from our units to kilograms, kilometers, seconds follow\n");
		fprintf(starStatsFile, " Mass in our units is the mass of an element. In other words the mass of an element is one.\n");
		fprintf(starStatsFile, " Length in our units is the starting diameter of an element. In other words the staring base diameter of an element is one.\n");
		fprintf(starStatsFile, " Time in our units is set so that the universal gravitational constant is 1.");
		fprintf(starStatsFile, "\n ");
		fprintf(starStatsFile, "\n Our length unit is this many kilometers: %e", SystemLengthConverterToKilometers);
		fprintf(starStatsFile, "\n Our mass unit is this many kilograms: %e", SystemMassConverterToKilograms);
		fprintf(starStatsFile, "\n Our time unit is this many seconds: %e or days %e", SystemTimeConverterToSeconds, SystemTimeConverterToSeconds/(60*60*24));
		fprintf(starStatsFile, "\n ");
		fprintf(starStatsFile, "\n Total number of elements in star1: %d", NumberOfElementsInStar1);
		fprintf(starStatsFile, "\n Total number of elements in star2: %d", NumberOfElementsInStar2);
		fprintf(starStatsFile, "\n ");
		fprintf(starStatsFile, "\n Mass of Star1 = %e kilograms", massStar1);
		fprintf(starStatsFile, "\n Radius of Star1 = %e kilometers", radiusStar1);
		fprintf(starStatsFile, "\n Density of star1 = %e kilograms/(cubic kilometer)", densityStar1);
		fprintf(starStatsFile, "\n ");
		fprintf(starStatsFile, "\n Mass of Star2 = %e kilograms", massStar2);
		fprintf(starStatsFile, "\n Radius of Star2 = %e kilometers", radiusStar2);
		fprintf(starStatsFile, "\n Density of star2 = %e kilograms/(cubic kilometer)", densityStar2);
	fclose(starStatsFile);
}

float nBody()
{ 
	float time;
	int   draw, record;
	float forceGrowth = 0.0;
	
	hipMemcpy(PosGPU, Pos, (NumberOfElements)*sizeof(float4), hipMemcpyHostToDevice);
	errorCheck("hipMemcpy Pos up");
	hipMemcpy(VelGPU, Vel, NumberOfElements*sizeof(float4), hipMemcpyHostToDevice);
	errorCheck("hipMemcpy Vel up");
	
	printf("\n************************************************** Simulation is on\n");
	time = 0.0;
	draw = 0;
	record = 0;
	while(time < BranchRunTime)
	{	
		getForce<<<GridConfig, BlockConfig>>>(PosGPU, VelGPU, ForceGPU, KH, KRH, forceGrowth, NumberOfElementsInStar1, (float)Pi);
		errorCheck("getForce");
		moveBodies<<<GridConfig, BlockConfig>>>(PosGPU, VelGPU, ForceGPU, Dt);
		errorCheck("moveBodiesDamped");
		
		if(GrowStartTime < time && time < GrowStopTime) 
		{
			forceGrowth += DeltaForceIncrease;
		}
		
		if(draw == DrawRate) 
		{
			hipMemcpy(Pos, PosGPU, (NumberOfElements)*sizeof(float4), hipMemcpyDeviceToHost);
			errorCheck("hipMemcpy Pos draw");
	    	drawPicture(Pos, NumberOfElementsInStar1, NumberOfElementsInStar2);
	    	printf("\n Time in days = %f\n", time*SystemTimeConverterToSeconds/(60.0*60.0*24.0));
			draw = 0;
		}
		draw++;
		
		if(record == RecordRate) 
		{
			hipMemcpy(Pos, PosGPU, (NumberOfElements)*sizeof(float4), hipMemcpyDeviceToHost);
			errorCheck("hipMemcpy Pos draw");
	    	recordPosAndVel();
			record = 0;
		}
		record++;
		
		time += Dt;
	}
	return(time -= Dt);
	//while(1);
}

static void signalHandler(int signum)
{
	int command;
    
	cout << "\n\n******************************************************" << endl;
	cout << "Enter:666 to kill the run." << endl;
	cout << "Enter:1 to change the draw rate." << endl;
	cout << "Enter:2 to continue the run." << endl;
	cout << "******************************************************\n\nCommand: ";
    
	cin >> command;
    
	if(command == 666)
	{
		cout << "\n\n******************************************************" << endl;
		cout << "Are you sure you want to terminate the run?" << endl;
		cout << "Enter:666 again if you are sure. Enter anything else to continue the run." << endl;
		cout << "******************************************************\n\nCommand: ";
		cin >> command;
		
		if(command == 666)
		{
			cleanUp();
			exit(0);
		}
	}
	else if(command == 1)
	{
		cout << "\nEnter the desired draw rate: ";
		cin >> DrawRate;
		cout << "\nDrawRate: " << DrawRate << endl;
	}
	else if (command == 2)
	{
		cout << "\nRun continued." << endl;
	}
	else
	{
		cout <<"\n\n Invalid Command\n" << endl;
	}
}

void control()
{	
	struct sigaction sa;
	float time;
	
	// Handling input from the screen.
	sa.sa_handler = signalHandler;
	sigemptyset(&sa.sa_mask);
	sa.sa_flags = SA_RESTART; // Restart functions if interrupted by handler
	if (sigaction(SIGINT, &sa, NULL) == -1)
	{
		printf("\nTSU Error: sigaction error\n");
	}

	// Creating folder to hold the newly created stars and moving into that folder. It also makes a copy of the BiuldSetup file in this folder.
	printf("\n Creating folders for the branch run.\n");
	createFolderForNewBranchRun();
	
	// Reading in the build parameters to a file.
	printf("\n Reading in the build parameters.\n");
	readBuildParameters();
	
	// Creating and saving the run the run parameters to a file.
	printf("\n Creating and saving the run parameters file.\n");
	generateRunParameters();
	
	// Allocating memory for CPU and GPU.
	printf("\n Allocating memory on the GPU and CPU and opening positions and velocities file.\n");
	allocateMemory();
	
	// Reading in the raw stars from the build generated by the build program.
	printf("\n Reading in the stars that were generated in the build probram.\n");
	readInTheInitialsStars();
	
	// Reading in Branch parameters and seting inintial conditions.
	printf("\n Reading in Branch parameters and seting inintial conditions.\n");
	readBranchParametersAndSetInitialConditions();
	
	// Seting up the GPU.
	printf("\n Setting up the GPU.\n");
	deviceSetup();
	
	// Running the simulation.
	printf("\n Running the simulation.\n");
	time = nBody();
	
	// Saving the the runs final positions and velosities.	
	printf("\n Saving the the runs final positions and velosities.\n");
	time = time*SystemTimeConverterToSeconds/(60.0*60.0*24.0); // Converting to hours.
	recordFinalPosVelStars(time);  
	
	// Saving any wanted stats about the run that you may want.
	printf("\n Saving any wanted stats about the run that you may want.\n");
	recordStarStats();	
	
	// Freeing memory. 	
	printf("\n Cleaning up the run.\n");
	cleanUp();

	printf("\n DONE \n");
	exit(0);
}

//Globals for viewing
//Viewing cropped pyrimid
double ViewBoxSize = 10.0;

GLdouble Left = -ViewBoxSize;
GLdouble Right = ViewBoxSize;
GLdouble Bottom = -ViewBoxSize;
GLdouble Top = ViewBoxSize;
GLdouble Front = ViewBoxSize;
GLdouble Back = -ViewBoxSize;

//Where your eye is located
GLdouble EyeX = 0.0;
GLdouble EyeY = 1.0;
GLdouble EyeZ = 1.0;

//Where you are looking
GLdouble CenterX = 0.0;
GLdouble CenterY = 0.0;
GLdouble CenterZ = 0.0;

//Up vector for viewing
GLdouble UpX = 0.0;
GLdouble UpY = 1.0;
GLdouble UpZ = 0.0;

void Display(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
}

void reshape(GLint w, GLint h) 
{
	glViewport(0, 0, w, h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
}

void init()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
}

int main(int argc, char** argv)
{ 
	if( argc < 2)
	{
		printf("\n You need to intire a root folder to work from on the comand line\n");
		exit(0);
	}
	else
	{
		strcat(RootFolderName, argv[1]);
	}

	//Globals for setting up the viewing window 
	int xWindowSize = 1500;
	int yWindowSize = 1500; 
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(xWindowSize,yWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Creating Stars");
	
	glutReshapeFunc(reshape);
	
	init();
	
	glShadeModel(GL_SMOOTH);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutIdleFunc(control);
	glutMainLoop();
	return 0;
}






